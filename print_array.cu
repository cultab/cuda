
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Print array, compiled for both host and device.
 */

#ifdef __CUDA_ARCH__
__device__
#else
__host__
#endif
void print_array(int *arr, size_t size, const char *name)
{
    printf("%s:\n", name);
    for (size_t i = 0; i < size; ++i) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}


#ifdef __CUDA_ARCH__
__device__
#else
__host__
#endif
void print_array(uint *arr, size_t size, const char *name)
{
    printf("%s:\n", name);
    for (size_t i = 0; i < size; ++i) {
        printf("%u ", arr[i]);
    }
    printf("\n");
}
