#include "hip/hip_runtime.h"
/* #include <hip/hip_runtime.h> */
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>

#include "types.h"
#include "print.h"

/*
 * Constant Definitions and Cuda weirdness
 */

// number of different keys to count
#define KEYS_COUNT 256

// we need at least KEYS_COUNT threads in total
#define THREADS 128
#define BLOCKS 20
#define SIZE 130

// HACK: parenthesis are VERY IMPORTANT
#define KEY_MAX_VALUE (KEYS_COUNT - 1)

#ifdef __CUDA_ARCH__
#define syncthreads() __syncthreads()
#else
#define syncthreads()
#endif

/*
 * Forward Declarations
 */

__global__ void count_atomic(elem *, size_t, uint *, uint, size_t);
__host__ uint *prefix_sum(uint *, size_t, int, int);
__global__ void prefix_sum_kernel(uint *, uint *, uint, size_t);
__global__ void move(uint *, elem *, uint, uint);

/*
 * Macros
 */

inline void cudaPrintError(hipError_t cudaerr, const char *file, int line)
{
    if (cudaerr != hipSuccess) {
        fprintf(stderr, "CUDA error: \"%s\" in file %s at line %d.\n", hipGetErrorString(cudaerr), file, line);
        exit(cudaerr);
    }
}

#define cudaErr(ans)                                                                                                   \
    do {                                                                                                               \
        cudaPrintError((ans), __FILE__, __LINE__);                                                                     \
    } while (0)

#define cudaLastErr()                                                                                                  \
    do {                                                                                                               \
        hipError_t cudaerr = hipDeviceSynchronize();                                                                 \
        cudaPrintError(cudaerr, __FILE__, __LINE__);                                                                   \
    } while (0)

/*
 * Functions
 */

__global__ void count_atomic(elem *array, size_t size, uint *counts, uint mask, size_t shift, size_t mask_size)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    __shared__ uint local_counts[KEYS_COUNT];

    if (tid < KEYS_COUNT) {
        local_counts[threadIdx.x] = 0;
    }
    /* printf("tid(%d) is here\n", tid); */
    syncthreads();

    for (size_t i = tid; i < size; i += stride) {
        /* debug("tid(%d): size=%ld stride=%d i=%ld\n", tid, size, stride, i); */
        atomicAdd(&local_counts[(array[i] & mask) >> (mask_size * shift)], 1);
        /* debug("tid(%d): local_counts[%u] = %d\n", tid, (array[i] & mask) >> (mask_size * shift), local_counts[(array[i] & mask) >> (mask_size * shift)]); */
    }

    /* __syncthreads(); if (tid == 0) print_array(local_counts, KEYS_COUNT, "local_counts"); */

    syncthreads();

    if (tid < KEYS_COUNT) {
        /* debug("adding local_counts[%d]=%d to counts[%d]=%d\n", threadIdx.x, local_counts[threadIdx.x], threadIdx.x, counts[threadIdx.x]); */
        atomicAdd(&(counts[threadIdx.x]), local_counts[threadIdx.x]);
    } else {
        /* debug("%d did nothing\n", tid); */
    }
}

/*
 * Implements All Partial Sums of an Array from:
 *
 *     Hillis, W. Daniel; Steele, Jr., Guy L. (December 1986).
 *     "Data parallel algorithms".
 *     Communications of the ACM. 29 (12): 1170–1183.
 *     doi:10.1145/7902.7903
 *
 */
 /*          = prefix_sum(      d_counts,  KEYS_COUNT,     blocks,     threads); */
__host__ uint *prefix_sum(uint *d_counts, size_t size, int blocks, int threads)
{
    uint *d_in;
    uint *d_out;
    uint *d_temp;

    /* uint *check = NULL; */
    /* check = (uint *)malloc(size * sizeof(uint)); */

    cudaErr(hipMalloc((void **)&d_out, size * sizeof(uint)));
    cudaErr(hipMalloc((void **)&d_in, size * sizeof(uint)));

    // initialize in and out array to counts
    cudaErr(hipMemcpy(d_in, d_counts, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToDevice));
    cudaErr(hipMemcpy(d_out, d_counts, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToDevice));

    for (int j = 1; j <= floor(log2(size)); j += 1) {
        prefix_sum_kernel<<<blocks, threads>>>(d_in, d_out, j, size);
        cudaLastErr();
        /* cudaErr(hipMemcpy(check, d_out, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToHost)); */
        /* print_array(check, size, "out array:"); */

        // copy result back to input
        cudaErr(hipMemcpy(d_in, d_out, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToDevice));
        // swap in and out
        d_temp = d_in;
        d_in = d_out;
        d_out = d_temp;
    }

    // free out
    cudaErr(hipFree(d_out));

    // NOTE: return input array (yes it's backwards)
    return d_in;
}

// TODO: maybe support ACTUALLY using multiple blocks
__global__ void prefix_sum_kernel(uint *in, uint *out, uint j, size_t size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // PERF: shift instead of pow(2, *)?
    // don't go out of bounds
    if (tid < size) {
        /* debug("tid(%d): something\n", tid); */
        /* printf("tid(%d) did something\n", tid); */
        if (tid >= __powf(2, j - 1)) {
            out[tid] += in[tid - (int)__powf(2, j - 1)];
            /* debug("out[%d] += %d\n", tid, in[tid - (int)__powf(2, j - 1)]); */
        }
    }
}

__global__ void hello()
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    debug("tid(%d): blockDim %d blockIdx %d threadIdx %d\n", tid, blockDim.x, blockIdx.x, threadIdx.x);
}

/* zero out a device array */
__global__ void zero_array(uint *d_array, size_t size) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (size_t i = tid; i < size; i += stride) {
        /* debug("write to %ld size = %ld\n", i, size); */
        d_array[i] = 0;
    }
}

int main(void)
{
    int threads = THREADS;
    int blocks = BLOCKS;
    size_t size = SIZE;

    if (threads * blocks < KEYS_COUNT) {
        printf("We need at least KEYS_COUNT(=%d) threads in total.\n", KEYS_COUNT);
        exit(-1);
    }

    elem *unsorted = NULL;
    elem *output = NULL;
    uint *counts = NULL;
    uint *prefix_sums = NULL;

    elem *d_unsorted = NULL;
    uint *d_counts = NULL;
    uint *d_prefix_sums = NULL;
    elem *d_output = NULL;

    unsorted = (elem *)malloc(size * sizeof(elem));
    if (unsorted == NULL) {/*{{{*/
        printf("malloc failed at line: %d in file %s\n", __LINE__, __FILE__);
        exit(-1);
    }/*}}}*/
    counts = (uint *)malloc(KEYS_COUNT * sizeof(uint));
    if (counts == NULL) {/*{{{*/
        printf("malloc failed at line: %d in file %s\n", __LINE__, __FILE__);
        exit(-1);
    }/*}}}*/
    prefix_sums = (uint *)malloc(KEYS_COUNT * sizeof(uint));
    if (prefix_sums == NULL) {/*{{{*/
        printf("malloc failed at line: %d in file %s\n", __LINE__, __FILE__);
        exit(-1);
    }/*}}}*/
    output = (elem *)malloc(size * sizeof(elem));
    if (output == NULL) {/*{{{*/
        printf("malloc failed at line: %d in file %s\n", __LINE__, __FILE__);
        exit(-1);
    }/*}}}*/

    cudaErr(hipMalloc((void **)&d_unsorted, size * sizeof(elem)));
    cudaErr(hipMalloc((void **)&d_counts, KEYS_COUNT * sizeof(uint)));
    cudaErr(hipMalloc((void **)&d_output, size * sizeof(elem)));

    for (size_t i = 0; i < size; ++i) {
        /* unsorted[i] = rand() % (1 * KEY_MAX_VALUE) + KEY_MAX_VALUE; */
        /* unsorted[i] = rand() % 1000; */
        /* unsorted[i] = rand(); */
        /* unsorted[i] = rand() % KEY_MAX_VALUE; */
        unsorted[i] = i;

        output[i] = -1337;
    }
    /* unsorted[2] = 0; */
    /* unsorted[4] = 1; */
    /* unsorted[5] = 255; */

    /* for (int i = 0; i < KEYS_COUNT; ++i) { */
    /*     counts[i] = 0; */
    /* } */

    // move array to device
    cudaErr(hipMemcpy(d_unsorted, unsorted, size * sizeof(elem), hipMemcpyHostToDevice));

    // for testing, to make sure the values change
    cudaErr(hipMemcpy(d_output, output, size * sizeof(elem), hipMemcpyHostToDevice));

    size_t elem_size = sizeof(elem) * 8;
    long unsigned iters = (long unsigned)((double)elem_size / log2(KEYS_COUNT));
    size_t mask_size = elem_size / iters;
    unsigned int mask = 0;
    unsigned int mask_shift = 0;
    /* unsigned int KEYS_COUNT = pow(2, 8); */

    for (size_t shift=0; shift < iters; ++shift) {

        zero_array<<<blocks, threads>>>(d_counts, KEYS_COUNT);
        cudaLastErr();

        debug("##########################\n# ITERATION %2lu OUT OF %2lu #\n##########################\n", shift+1, iters);
        // keep a copy of the mask
        uint old_mask = mask;

        // create a mask of size mask_s
        for (; mask_shift < mask_size * (shift + 1); mask_shift++) {
                mask |= (1 << mask_shift);
        }

        // TODO: comments
        // now use the old_mask to trim off bits that we already used
        if (shift > 0) {
            mask ^= old_mask;
        }

        debug("mask:\n");
        print_bits(mask);

        print_array(unsorted, size, "unsorted");
        /* print_array_bits(unsorted, size, "unsorted bits"); */

        // count frequencies
        count_atomic<<<blocks, threads>>>(d_unsorted, size, d_counts, mask, shift, mask_size);
        cudaLastErr();

        // copy counts back to host only to print them
        cudaErr(hipMemcpy(counts, d_counts, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToHost));
        print_array(counts, KEYS_COUNT, "counts");

        // get prefix sums of counts
        d_prefix_sums = prefix_sum(d_counts, KEYS_COUNT, blocks, threads);

        // copy prefix sums back to host because we need them
        cudaErr(hipMemcpy(prefix_sums, d_prefix_sums, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToHost));
        print_array(prefix_sums, KEYS_COUNT, "prefix_sum");
        /* print_compare_array(counts, prefix_sums, KEYS_COUNT); */

        /* move elements to sorted position */
        int offset = 0;
        prefix_sums[KEYS_COUNT - 1] = 0;
        for (size_t j = 0; j < size; ++j) {
            unsigned long masked_elem = (unsorted[j] & mask) >> (mask_size * shift);
            
            /* printf("elem %d\nmasked ", unsorted[j]); */
            /* print_bits(masked_elem); */

            if (masked_elem != 0) {
                offset = prefix_sums[masked_elem - 1];
                /* debug("! offset = prefix_sums[%lu] = %d, elem = %d, masked = %lu\n", masked_elem - 1, offset, unsorted[j], masked_elem); */
                prefix_sums[masked_elem - 1] += 1;
                /* debug("n moved unsorted[%4lu]=%4d to output[%4d]\n", j, unsorted[j], offset); */
            } else {
                offset = prefix_sums[KEYS_COUNT - 1];
                /* debug("0 offset = prefix_sums[%d] = %d, elem = %d, masked = %lu\n", KEYS_COUNT - 1, offset, unsorted[j], masked_elem); */
                prefix_sums[KEYS_COUNT - 1] += 1;
                /* debug("0 moved unsorted[%4lu]=%4d to output[%4d]\n", j, unsorted[j], offset); */
            }

            /* if (offset > size) { */
            /*     debug("OFFSET = %d mskelem = %lu\n", offset, masked_elem); */
            /*     exit(-1); */
            /* } */

            output[offset] = unsorted[j];
        }

        print_array(output, size, "sorted");
        /* print_array_bits(output, size, "sorted bits"); */

        cudaErr(hipMemcpy(d_unsorted, output, size * sizeof(elem), hipMemcpyHostToDevice));
        cudaErr(hipMemcpy(unsorted, output, size * sizeof(elem), hipMemcpyHostToHost));
    }

    /* free device memory */
    puts("FREE DEVICE");
    cudaErr(hipFree((void*)d_unsorted));
    cudaErr(hipFree((void*)d_counts));
    cudaErr(hipFree((void*)d_prefix_sums));
    cudaErr(hipFree((void*)d_output));

    /* free host memory */
    puts("FREE HOST");
    free(unsorted);
    free(counts);
    free(prefix_sums);
    free(output);

    puts("DONE");
}

/* trash code {{{ */

/* nope */
/* __global__ void move(elem *unsorted, size_t size, uint *prefix_sums, elem *output, uint mask, uint shift) */
/* { */
/*     int tid = blockDim.x * blockIdx.x + threadIdx.x; */
/*     int stride = blockDim.x * gridDim.x; */
/*     int offset = 0; */
/*  */
    /*     __shared__ uint local_offsets[KEYS_COUNT]; */
/*  */
    /* // offset is prefix sum of previous number, */
    /* // if there is no previous thread, use the last pos in the array, */
    /* // initializing it to zero */
    /* if (tid == 0) { */
    /*     prefix_sums[KEYS_COUNT - 1] = prefix_sums[1]; */
    /* } */
/*  */
/*     syncthreads(); */
/*  */
/*     // i is int, should it be size_t? */
/*     for (int i = size - tid - 1; i >= 0; i -= stride) { */
/*         if ((unsorted[i] & mask) >> (8 * shift) != 0) { */
/*             offset = atomicSub(&prefix_sums[(unsorted[i] & mask) >> (8 * shift)], 1); */
/*             debug("tid(%d) move unsorted[%d]=%d to output[%d]=%d\n", tid, i, unsorted[i], offset - 1, output[offset - 1]); */
/*             output[offset - 1] = unsorted[i]; */
/*         } */
/*     } */
/*  */
/*     syncthreads(); */
/*  */
/*  */
    /* __syncthreads(); */
    /* if (tid == 0) print_array(local_counts, KEYS_COUNT, "local_counts"); */
/* } */

/*         prefix_sums[KEYS_COUNT - 1] = 0; */
/*  */
/*         for (int j = (int)size - 1; j >= 0; --j) { */
/*             unsigned long masked_elem = (unsorted[j] & mask) >> (mask_size * shift); */
/*              */
/*             printf("elem %d\nmasked ", unsorted[j]); */
/*             print_bits(masked_elem); */
/*  */
/*             if (masked_elem != 0) { */
/*                 offset = prefix_sums[masked_elem - 1]; */
                /* debug("! offset = prefix_sums[%lu] = %d, elem = %d, masked = %lu\n", masked_elem - 1, offset, unsorted[j], masked_elem); */
/*                 prefix_sums[masked_elem - 1] += 1; */
/*             } else { */
/*                 offset = prefix_sums[KEYS_COUNT - 1]; */
                /* debug("0 offset = prefix_sums[%d] = %d, elem = %d, masked = %lu\n", KEYS_COUNT - 1, offset, unsorted[j], masked_elem); */
/*                 prefix_sums[KEYS_COUNT - 1] += 1; */
/*             } */
/*  */
            /* if (offset > size) { */
            /*     debug("OFFSET = %d mskelem = %lu\n", offset, masked_elem); */
            /*     exit(-1); */
            /* } */
/*  */
/*             debug("moved unsorted[%4d]=%4d to output[%4d]\n", j, unsorted[j], offset); */
/*             output[offset] = unsorted[j]; */
/*         } */

        /* prefix_sums[KEYS_COUNT - 1] = prefix_sums[0] - 1; */



// }}}

/* commented  out code {{{*/
/* __global__ void move(int *array, int size, int *prefix, int *output, unsigned int mask) { */
/*         int tid = threadIdx.x; */
/*         int offset = 0; */
/*  */
/*         if (tid != 0) { */
/*                 offset = prefix[tid - 1]; */
/*         } else { */
/*                 offset = 0; */
/*                 // print_arr_in_gpu(prefix, size, "dev_prefix"); */
/*                 // printf("-------------- prefix[tid-1] = prefix[%d] = %d\n", tid-1, prefix[tid-1]); */
/*                 // printf("-------------- prefix[tid-2] = prefix[%d] = %d\n", tid-2, prefix[tid-2]); */
/*         } */
/*  */
/*         printf("tid(%d): offset=%d\n", tid, offset); */
/*  */
/*         for (int i=size-1; i>=0; --i) { */
/*                 // if this thread cares for the current number */
/*                 if ((array[i]) == tid) { */
/*                         output[offset++] = array[i]; */
/*                         printf("moving %d from array[%d] to output[%d]\n", array[i], i, offset - 1); */
/*                         // printf("output[%d] = array[%d] = %d\n", offset - 1, i, array[i]); */
/*                 } */
/*         } */
/* } */
/*}}}*/
