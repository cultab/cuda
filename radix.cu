#include "hip/hip_runtime.h"
/* #include <hip/hip_runtime.h> */
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <set>

#include "types.cuh"
#include "print.cuh"

/*
 * Constant Definitions and Cuda weirdness
 */

// number of different keys to count, must be 2^n where n is a power of 2
// #define KEYS_COUNT 4096

/*
#define KEYS_COUNT 2    // 2 ^ 1
#define KEYS_COUNT 4    // 2 ^ 2
#define KEYS_COUNT 16   // 2 ^ 4 
*/
#define KEYS_COUNT 256  // 2 ^ 8, just right ?
/*
#define KEYS_COUNT 6536 // 2 ^ 16, way too big
*/

// we need at least KEYS_COUNT threads in total
#define THREADS 128
// multiple of 96 for 1050ti says nvvp
#define BLOCKS (96 * 2)
// size of unsorted array
#define SIZE 400000000
// #define SIZE 10
// #define PRINT

// HACK: parenthesis are VERY IMPORTANT
#define KEY_MAX_VALUE (KEYS_COUNT - 1)

#ifdef __CUDA_ARCH__
#define syncthreads() __syncthreads()
#else
#define syncthreads()
#endif

/*
 * Forward Declarations
 */

__global__ void count_atomic(elem *, size_t, uint *, uint, size_t);
__host__ uint *prefix_sum(uint *, size_t, int, int);
__global__ void prefix_sum_kernel(uint *, uint *, uint, size_t);
__global__ void move(elem *, size_t, uint *, elem, uint, uint, size_t, size_t);
__host__ void host_move(uint *, elem *, elem *, size_t, uint, ulong, ulong);

/*
 * Macros
 */

inline void cudaPrintError(hipError_t cudaerr, const char *file, int line)
{
    if (cudaerr != hipSuccess) {
        fprintf(stderr, "CUDA error: \"%s\" in file %s at line %d.\n", hipGetErrorString(cudaerr), file, line);
        exit(cudaerr);
    }
}

#define cudaErr(ans)                                                                                                   \
    do {                                                                                                               \
        cudaPrintError((ans), __FILE__, __LINE__);                                                                     \
    } while (0)

#define cudaLastErr()                                                                                                  \
    do {                                                                                                               \
        hipError_t cudaerr = hipDeviceSynchronize();                                                                 \
        cudaPrintError(cudaerr, __FILE__, __LINE__);                                                                   \
    } while (0)

/*
 * Functions
 */

__global__ void count_atomic(elem *array, size_t size, uint *counts, uint mask, size_t shift, size_t mask_size)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    /* block id and stride */
    int block_tid = threadIdx.x;
    int block_stride = blockDim.x;

    __shared__ uint local_counts[KEYS_COUNT];

    // zero out the block local shared memory
    for (size_t i = block_tid; i < KEYS_COUNT; i += block_stride) {
        local_counts[i] = 0;
        // debug("block(%d) zero'ed [%lu]\n", blockIdx.x, i);
    }
    // printf("tid(%d) is here\n", tid);
    syncthreads();

    // if (tid == 0) {
    //     print_array(counts, KEYS_COUNT, "local_counts -----------");
    // }
    // syncthreads();

    for (size_t i = tid; i < size; i += stride) {
        // debug("tid(%d): size=%ld stride=%d i=%ld\n", tid, size, stride, i);
        atomicAdd(&local_counts[(array[i] & mask) >> (mask_size * shift)], 1);
        // debug("tid(%d): local_counts[%u] = %d\n", tid, (array[i] & mask) >> (mask_size * shift), local_counts[(array[i] & mask) >> (mask_size * shift)]);
    }

    // __syncthreads(); if (tid == 0) print_array(local_counts, KEYS_COUNT, "local_counts");

    syncthreads();

    // copy per block results back to global memory
    for (size_t i = block_tid; i < KEYS_COUNT; i += block_stride) {
        // debug("block(%d): adding local_counts[%lu]=%u to counts[%lu]=%u\n",blockIdx.x, i, local_counts[i], i, counts[i]);
        atomicAdd(&(counts[i]), local_counts[i]);
    } 
}

/*
 * Implements All Partial Sums of an Array from:
 *
 *     Hillis, W. Daniel; Steele, Jr., Guy L. (December 1986).
 *     "Data parallel algorithms".
 *     Communications of the ACM. 29 (12): 1170–1183.
 *     doi:10.1145/7902.7903
 *
 */
 /*          = prefix_sum(      d_counts,  KEYS_COUNT,     blocks,     threads); */
__host__ uint *prefix_sum(uint *d_counts, size_t size, int blocks, int threads)
{
    uint *d_in;
    uint *d_out;
    uint *d_temp;

    // uint *check = NULL;
    // check = (uint *)malloc(size * sizeof(uint));

    cudaErr(hipMalloc((void **)&d_out, size * sizeof(uint)));
    cudaErr(hipMalloc((void **)&d_in, size * sizeof(uint)));

    // initialize in and out array to counts
    cudaErr(hipMemcpy(d_in, d_counts, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToDevice));
    cudaErr(hipMemcpy(d_out, d_counts, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToDevice));

    for (int j = 1; j <= floor(log2(size)); j += 1) {
        prefix_sum_kernel<<<blocks, threads>>>(d_in, d_out, j, size);
        cudaLastErr();
        // cudaErr(hipMemcpy(check, d_out, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToHost));
        // print_array(check, size, "out array:");

        // copy result back to input
        cudaErr(hipMemcpy(d_in, d_out, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToDevice));
        // swap in and out
        d_temp = d_in;
        d_in = d_out;
        d_out = d_temp;
    }

    // free out
    cudaErr(hipFree(d_out));

    // NOTE: return input array (yes it's backwards)
    return d_in;
}

// TODO: maybe support ACTUALLY using multiple blocks
__global__ void prefix_sum_kernel(uint *in, uint *out, uint j, size_t size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // PERF: shift instead of pow(2, *)?
    // don't go out of bounds
    if (tid < size) {
        // debug("tid(%d): something\n", tid);
        // printf("tid(%d) did something\n", tid);
        if (tid >= __powf(2, j - 1)) {
            out[tid] += in[tid - (int)__powf(2, j - 1)];
            // debug("out[%d] += %d\n", tid, in[tid - (int)__powf(2, j - 1)]);
        }
    }
}

__global__ void hello()
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    debug("tid(%d): blockDim %d blockIdx %d threadIdx %d\n", tid, blockDim.x, blockIdx.x, threadIdx.x);
}

/* zero out a device array */
__global__ void zero_array(uint *d_array, size_t size) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (size_t i = tid; i < size; i += stride) {
        // debug("write to %ld size = %ld\n", i, size);
        d_array[i] = 0;
    }
}

/*
 * Should be called with KEYS_COUNT many threads in total.
 * Assuming 256 KEYS_COUNT and at least 32 threads per block so we get a full warp,
 * 8 blocks with 32 threads each seems reasonable.
 *
 * Allows us to possibly not copy between host and device so much.
 *
 */
__global__ void move(elem *d_unsorted, size_t size, uint *d_prefix, elem *d_sorted, uint mask, size_t shift, size_t mask_size) {
    // use the thread id as a mask since we launch KEYS_COUNT many threads
    int tid = blockDim.x * blockIdx.x + threadIdx.x;


    int offset = 0;

    if (tid == 0) {
        d_prefix[KEYS_COUNT - 1] = 0;
    }
    // no need to sync since tid == 0 only will write to this address

    for (size_t j = 0; j < size; ++j) {
        ulong masked_elem = (d_unsorted[j] & mask) >> (mask_size * shift);

        // if this thread should handle this element
        if (masked_elem == tid) {
            if (masked_elem != 0) {
                offset = d_prefix[masked_elem - 1];
                d_prefix[masked_elem - 1] += 1;
            } else {
                offset = d_prefix[KEYS_COUNT - 1];
                d_prefix[KEYS_COUNT - 1] += 1;
            }
            // debug("tid(%d) moved unsorted[%4lu]=%4d(%4d) to output[%4d]\n", j, d_unsorted[j], masked_elem, offset);
            d_sorted[offset] = d_unsorted[j];
        }
    }
}


void host_move(uint *prefix_sums, elem *unsorted, elem *sorted, size_t size, uint mask, ulong mask_size, ulong shift) {

    /* move elements to sorted position *//*{{{*/
    int offset = 0;
    prefix_sums[KEYS_COUNT - 1] = 0;
    // prefix_sums[KEYS_COUNT - 1] = prefix_sums[0];
    for (size_t j = 0; j < size; ++j) {
        ulong masked_elem = (unsorted[j] & mask) >> (mask_size * shift);

        if (masked_elem != 0) {
            offset = prefix_sums[masked_elem - 1];
            // debug("! offset = prefix_sums[%lu] = %d, elem = %d, masked = %lu\n", masked_elem - 1, offset, unsorted[j], masked_elem);
            prefix_sums[masked_elem - 1] += 1;
            // debug("n moved unsorted[%4lu]=%4d(%4d) to output[%4d]\n", j, unsorted[j], masked_elem, offset);
        } else {
            offset = prefix_sums[KEYS_COUNT - 1];
            // debug("0 offset = prefix_sums[%d] = %d, elem = %d, masked = %lu\n", KEYS_COUNT - 1, offset, unsorted[j], masked_elem);
            prefix_sums[KEYS_COUNT - 1] += 1;
            // debug("0 moved unsorted[%4lu]=%4d(%4d) to output[%4d]\n", j, unsorted[j], masked_elem, offset);
        }
        sorted[offset] = unsorted[j];
    }

    // print_array(output, size, "sorted");
    // print_array_bits(output, size, "sorted bits");

    // if (shift == 0) {
    //     exit(0);
    // }}}}
}


int main(void)
{
    printf("Start!\n");
    float time;
    int threads = THREADS;
    int blocks = BLOCKS;
    size_t size = SIZE;

    // needed for current prefix_sum implementation
    if (threads * blocks < KEYS_COUNT) {
        fprintf(stderr, "We need at least KEYS_COUNT(=%d) threads in total. We have %d..\n", KEYS_COUNT, threads * blocks);
        exit(-1);
    }

    hipEvent_t start, stop;

    cudaErr(hipEventCreate(&start));
    cudaErr(hipEventCreate(&stop));

    elem *unsorted = NULL;
    elem *sorted = NULL;
    uint *counts = NULL;
    uint *prefix_sums = NULL;

    elem *d_unsorted = NULL;
    uint *d_counts = NULL;
    uint *d_prefix_sums = NULL;
    elem *d_sorted = NULL;

    printf("Allocating memory for unsorted.\n");
    unsorted = (elem *)malloc(size * sizeof(elem));
    if (unsorted == NULL) {/*{{{*/
        printf("malloc failed at line: %d in file %s\n", __LINE__, __FILE__);
        exit(-1);
    }/*}}}*/
    printf("Allocating memory for counts.\n");
    counts = (uint *)malloc(KEYS_COUNT * sizeof(uint));
    if (counts == NULL) {/*{{{*/
        printf("malloc failed at line: %d in file %s\n", __LINE__, __FILE__);
        exit(-1);
    }/*}}}*/
    printf("Allocating memory for prefix_sums.\n");
    prefix_sums = (uint *)malloc(KEYS_COUNT * sizeof(uint));
    if (prefix_sums == NULL) {/*{{{*/
        printf("malloc failed at line: %d in file %s\n", __LINE__, __FILE__);
        exit(-1);
    }/*}}}*/
    printf("Allocating memory for sorted.\n");
    sorted = (elem *)malloc(size * sizeof(elem));
    if (sorted == NULL) {/*{{{*/
        printf("malloc failed at line: %d in file %s\n", __LINE__, __FILE__);
        exit(-1);
    }/*}}}*/


    printf("Allocating memory for d_unsorted.\n");
    cudaErr(hipMalloc((void **)&d_unsorted, size * sizeof(elem)));
    printf("Allocating memory for d_counts.\n");
    cudaErr(hipMalloc((void **)&d_counts, KEYS_COUNT * sizeof(uint)));
    printf("Allocating memory for d_sorted.\n");
    cudaErr(hipMalloc((void **)&d_sorted, size * sizeof(elem)));

    printf("Generating input array.\n");
    for (size_t i = 0; i < size; ++i) {
        unsorted[i] = rand() % INT32_MAX;
        // unsorted[i] = rand() % (1 * KEY_MAX_VALUE) + KEY_MAX_VALUE;
        // unsorted[i] = rand() % 1000;
        // unsorted[i] = rand();
        // unsorted[i] = rand() % 10000;
        // unsorted[i] = rand() % KEY_MAX_VALUE;
        // unsorted[i] = size - i;

        sorted[i] = -1337;
    }

    // move array to device
    cudaErr(hipMemcpy(d_unsorted, unsorted, size * sizeof(elem), hipMemcpyHostToDevice));

    // for testing, to make sure the values change
    // cudaErr(hipMemcpy(d_output, output, size * sizeof(elem), hipMemcpyHostToDevice));

    // size of elem in bits
    size_t elem_bit_size = sizeof(elem) * 8;
    // number of iterations needed to sort based on all bits
    ulong iters = (ulong)((double)elem_bit_size / log2(KEYS_COUNT));
    // size of the mask used to extract a key from an elem
    size_t mask_size = elem_bit_size / iters;

    uint mask = 0;
    uint mask_shift = 0;

    printf("KEYS_COUNT=%d\n", KEYS_COUNT);
    printf("elem_size=%lu\n", elem_bit_size);
    printf("iters=%lu\n", iters);
    printf("mask_size=%lu\n", mask_size);

    // check if KEYS_COUNT is correctly set
    if (mask_size * iters != elem_bit_size) {
        fprintf(stderr, "Bad KEYS_COUNT=%d value, elem_size=%lu is not integer "
            "divisible into iters=%lu many parts.\n", KEYS_COUNT, elem_bit_size, iters);
        exit(-1);
    }

    // print_array(unsorted, size, "unsorted");

    // record start time
    hipEventRecord(start);

    for (size_t shift=0; shift < iters; ++shift) {

        printf("In device zero_array()\n");
        zero_array<<<blocks, threads>>>(d_counts, KEYS_COUNT);
        cudaLastErr();
        printf("Out of device zero_array()\n");

        debug("##########################\n# ITERATION %2lu OUT OF %2lu #\n##########################\n", shift+1, iters);
        // keep a copy of the mask
        uint old_mask = mask;

        // create a mask of size mask_size shifted appropriately
        for (; mask_shift < mask_size * (shift + 1); mask_shift++) {
                mask |= (1 << mask_shift);
        }

        // TODO: comments
        // now use the old_mask to trim off bits that we already used
        if (shift > 0) {
            mask ^= old_mask;
        }

        debug("mask:\n");
        print_bits(mask);

        // print_array(unsorted, size, "unsorted");
        // print_array_bits(unsorted, size, "unsorted bits");

        printf("In device count_atomic()\n");
        // count frequencies
        count_atomic<<<blocks, threads>>>(d_unsorted, size, d_counts, mask, shift, mask_size);
        cudaLastErr();
        printf("Out of device zero_array()\n");

        // copy counts back to host only to print them
        // cudaErr(hipMemcpy(counts, d_counts, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToHost));
        // print_array(counts, KEYS_COUNT, "counts");

        printf("In prefix_sum()\n");
        // get prefix sums of counts
        d_prefix_sums = prefix_sum(d_counts, KEYS_COUNT, blocks, threads);
        printf("Out of prefix_sum()\n");

        // copy prefix sums back to host because we *might* need them
        cudaErr(hipMemcpy(prefix_sums, d_prefix_sums, KEYS_COUNT * sizeof(uint), hipMemcpyDeviceToHost));
        // print_array(prefix_sums, KEYS_COUNT, "prefix_sum");
        // print_compare_array(counts, prefix_sums, KEYS_COUNT);

        printf("In host move()\n");
        host_move(prefix_sums, unsorted, sorted, size, mask, mask_size, shift);
        printf("Out host of move()\n");
        cudaErr(hipMemcpy(d_unsorted, sorted, size * sizeof(elem), hipMemcpyHostToDevice));
        memcpy(unsorted, sorted, size * sizeof(elem));

        // printf("In device move()\n");
        // move<<<8, 32>>>(d_unsorted, size, d_prefix_sums, d_sorted, mask, shift, mask_size);
        // cudaLastErr();
        // printf("Out of device move()\n");
        // cudaErr(hipMemcpy(d_unsorted, d_sorted, size * sizeof(elem), hipMemcpyDeviceToDevice));

        // cudaErr(hipMemcpy(unsorted, d_sorted, size * sizeof(elem), hipMemcpyDeviceToHost));

        // print_array(sorted, size, "sorted");
    }
    // cudaErr(hipMemcpy(sorted, d_sorted, size * sizeof(elem), hipMemcpyDeviceToHost));

    // record stop time
    cudaErr(hipEventRecord(stop));

    hipEventElapsedTime(&time, start, stop);

    printf("Finished sorting in %.2f ms!\n", time);
    #ifdef PRINT
    print_array(sorted, size, "sorted");
    #endif

    /* free device memory */
    puts("Free device memory.");
    cudaErr(hipFree((void*)d_unsorted));
    cudaErr(hipFree((void*)d_counts));
    cudaErr(hipFree((void*)d_prefix_sums));
    cudaErr(hipFree((void*)d_sorted));

    /* free host memory */
    puts("Free host memory.");
    free(unsorted);
    free(counts);
    free(prefix_sums);
    free(sorted);

    puts("DONE");
}

/* trash code {{{ */

/* nope */
/* __global__ void move(elem *unsorted, size_t size, uint *prefix_sums, elem *output, uint mask, uint shift) */
/* { */
/*     int tid = blockDim.x * blockIdx.x + threadIdx.x; */
/*     int stride = blockDim.x * gridDim.x; */
/*     int offset = 0; */
/*  */
    /*     __shared__ uint local_offsets[KEYS_COUNT]; */
/*  */
    /* // offset is prefix sum of previous number, */
    /* // if there is no previous thread, use the last pos in the array, */
    /* // initializing it to zero */
    /* if (tid == 0) { */
    /*     prefix_sums[KEYS_COUNT - 1] = prefix_sums[1]; */
    /* } */
/*  */
/*     syncthreads(); */
/*  */
/*     // i is int, should it be size_t? */
/*     for (int i = size - tid - 1; i >= 0; i -= stride) { */
/*         if ((unsorted[i] & mask) >> (8 * shift) != 0) { */
/*             offset = atomicSub(&prefix_sums[(unsorted[i] & mask) >> (8 * shift)], 1); */
/*             debug("tid(%d) move unsorted[%d]=%d to output[%d]=%d\n", tid, i, unsorted[i], offset - 1, output[offset - 1]); */
/*             output[offset - 1] = unsorted[i]; */
/*         } */
/*     } */
/*  */
/*     syncthreads(); */
/*  */
/*  */
    /* __syncthreads(); */
    /* if (tid == 0) print_array(local_counts, KEYS_COUNT, "local_counts"); */
/* } */

/*         prefix_sums[KEYS_COUNT - 1] = 0; */
/*  */
/*         for (int j = (int)size - 1; j >= 0; --j) { */
/*             ulong masked_elem = (unsorted[j] & mask) >> (mask_size * shift); */
/*              */
/*             printf("elem %d\nmasked ", unsorted[j]); */
/*             print_bits(masked_elem); */
/*  */
/*             if (masked_elem != 0) { */
/*                 offset = prefix_sums[masked_elem - 1]; */
                /* debug("! offset = prefix_sums[%lu] = %d, elem = %d, masked = %lu\n", masked_elem - 1, offset, unsorted[j], masked_elem); */
/*                 prefix_sums[masked_elem - 1] += 1; */
/*             } else { */
/*                 offset = prefix_sums[KEYS_COUNT - 1]; */
                /* debug("0 offset = prefix_sums[%d] = %d, elem = %d, masked = %lu\n", KEYS_COUNT - 1, offset, unsorted[j], masked_elem); */
/*                 prefix_sums[KEYS_COUNT - 1] += 1; */
/*             } */
/*  */
            /* if (offset > size) { */
            /*     debug("OFFSET = %d mskelem = %lu\n", offset, masked_elem); */
            /*     exit(-1); */
            /* } */
/*  */
/*             debug("moved unsorted[%4d]=%4d to output[%4d]\n", j, unsorted[j], offset); */
/*             output[offset] = unsorted[j]; */
/*         } */

        /* prefix_sums[KEYS_COUNT - 1] = prefix_sums[0] - 1; */



// }}}

/* commented  out code {{{*/
/* __global__ void move(int *array, int size, int *prefix, int *output, uint mask) { */
/*         int tid = threadIdx.x; */
/*         int offset = 0; */
/*  */
/*         if (tid != 0) { */
/*                 offset = prefix[tid - 1]; */
/*         } else { */
/*                 offset = 0; */
/*                 // print_arr_in_gpu(prefix, size, "dev_prefix"); */
/*                 // printf("-------------- prefix[tid-1] = prefix[%d] = %d\n", tid-1, prefix[tid-1]); */
/*                 // printf("-------------- prefix[tid-2] = prefix[%d] = %d\n", tid-2, prefix[tid-2]); */
/*         } */
/*  */
/*         printf("tid(%d): offset=%d\n", tid, offset); */
/*  */
/*         for (int i=size-1; i>=0; --i) { */
/*                 // if this thread cares for the current number */
/*                 if ((array[i]) == tid) { */
/*                         output[offset++] = array[i]; */
/*                         printf("moving %d from array[%d] to output[%d]\n", array[i], i, offset - 1); */
/*                         // printf("output[%d] = array[%d] = %d\n", offset - 1, i, array[i]); */
/*                 } */
/*         } */
/* } */
/*}}}*/
