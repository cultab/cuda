#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>

#include "types.h"
#include "print_array.h"

/*
 * Type and Constant Definitions
 */

// number of different keys to count
#define MAGIC_NUM 256

/*
 * Forward Declarations
 */

__global__ void countAtomic(elem *, int, uint *);

__host__ uint *prefix_sum(uint *, size_t, int, int);
__global__ void prefix_sum_kernel(uint *, uint *, uint, size_t);

__global__ void move(uint *, elem *);

/*
 * Macros
 */

inline void cudaPrintError(hipError_t cudaerr, char *file, int line)
{
    if (cudaerr != hipSuccess) {
        fprintf(stderr, "CUDA error: \"%s\" in file %s at line %d.\n", hipGetErrorString(cudaerr), __FILE__, __LINE__);
        exit(cudaerr);
    }
}

#define cudaErr(ans)                                                                                                   \
    do {                                                                                                               \
        cudaPrintError((ans), __FILE__, __LINE__);                                                                     \
    } while (0)

#define cudaLastErr()                                                                                                  \
    do {                                                                                                               \
        hipError_t cudaerr = hipDeviceSynchronize();                                                                 \
        cudaPrintError(cudaerr, __FILE__, __LINE__);                                                                   \
    } while (0)

__global__ void test()
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    printf("%d here\n", tid);
}
__global__ void countAtomic(elem *array, int size, uint *counts)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    __shared__ uint local_counts[MAGIC_NUM];

    if (tid < MAGIC_NUM) {
        local_counts[threadIdx.x] = 0;
    }
    __syncthreads();

    // HACK: make it by order
    static uint mask = 0b11111111;

    for (size_t i = tid; i < size; i += stride) {
        /* printf("tid(%d): size=%d stride=%d i=%lu\n", tid, size, stride, i); */
        /* printf("%lu %p\n", i, array); */
        atomicAdd(&local_counts[array[i] & mask], 1);
        printf("tid(%d): local_counts[%u] = %d\n", tid, array[i] & mask, local_counts[array[i] & mask]);
    }

    /* __syncthreads(); */
    /* if (tid == 0) print_array(local_counts, MAGIC_NUM, "local_counts"); */

    __syncthreads();

    if (tid < MAGIC_NUM) {
        /* printf("adding local_counts[%d]=%d to counts[%d]=%d\n", threadIdx.x, local_counts[threadIdx.x], threadIdx.x, */
        /* counts[threadIdx.x]); */
        atomicAdd(&(counts[threadIdx.x]), local_counts[threadIdx.x]);
    } else {
        /* printf("%d did nothing\n", tid); */
    }
}

__host__ uint *prefix_sum(uint *counts, size_t size, int blocks, int threads)
{
    uint *d_in;
    uint *d_out;
    uint *d_temp;

    /* uint *check = NULL; */
    /* check = (uint *)malloc(size * sizeof(uint)); */

    cudaErr(hipMalloc((void **)&d_out, size * sizeof(uint)));
    cudaErr(hipMalloc((void **)&d_in, size * sizeof(uint)));

    // initialize in and out array to counts
    cudaErr(hipMemcpy(d_in, counts, MAGIC_NUM * sizeof(uint), hipMemcpyDeviceToDevice));
    cudaErr(hipMemcpy(d_out, counts, MAGIC_NUM * sizeof(uint), hipMemcpyDeviceToDevice));

    for (int j = 1; j <= floor(log2(size)); j += 1) {
        prefix_sum_kernel<<<blocks, threads>>>(d_in, d_out, j, size);
        cudaLastErr();

        /* cudaErr(hipMemcpy(check, d_out, MAGIC_NUM * sizeof(uint), hipMemcpyDeviceToHost)); */
        /* print_array(check, size, "out array:"); */
        // copy result back to input
        cudaErr(hipMemcpy(d_in, d_out, MAGIC_NUM * sizeof(uint), hipMemcpyDeviceToDevice));
        // swap in and out
        d_temp = d_in;
        d_in = d_out;
        d_out = d_temp;
    }

    // free out
    cudaErr(hipFree(d_out));

    // return input array (yes)
    return d_in;
}

// TODO: maybe support ACTUALLY using multiple blocks
__global__ void prefix_sum_kernel(uint *in, uint *out, uint j, size_t size)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // don't go out of bounds
    if (tid < size) {
        if (tid >= pow(2, j - 1)) {
            /* printf("adding %d to %d\n",in_idx, out_idx); */
            out[tid] += in[tid - (int)pow(2, j - 1)];
        }
    }
}

__global__ void move(elem *unsorted, size_t size, uint *prefix_sums, elem *output)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int offset = 0;

    /*     __shared__ uint local_offsets[MAGIC_NUM]; */
    /*  */
    /*     if (tid < MAGIC_NUM) { */
    /*         local_offsets[threadIdx.x] = 0; */
    /*     } */
    /*     __syncthreads(); */

    // HACK: make it by order
    static uint mask = 0b11111111;

    // offset is prefix sum of previous number,
    // if there is no previous thread, use the last pos in the array,
    // initializing it to zero
    if (tid == 0) {
        prefix_sums[MAGIC_NUM - 1] = 0;
    }

    // i is int, should it be size_t?
    for (int i = size - tid - 1; i >= 0; i -= stride) {
        if (unsorted[i] == 0) {
            offset = atomicAdd(&prefix_sums[MAGIC_NUM - 1], 1);
        } else {
            offset = atomicAdd(&prefix_sums[unsorted[i] - 1], 1);
        }
        printf("tid(%d) move unsorted[%d]=%d to output[%d]=%d\n", tid, i, unsorted[i], offset, output[offset]);
        output[offset] = unsorted[i];
    }

    /* __syncthreads(); */
    /* if (tid == 0) print_array(local_counts, MAGIC_NUM, "local_counts"); */

    /* __syncthreads(); */
}

/* __global__ void move(int *array, int size, int *prefix, int *output, unsigned int mask) { */
/*         int tid = threadIdx.x; */
/*         int offset = 0; */
/*  */
/*         if (tid != 0) { */
/*                 offset = prefix[tid - 1]; */
/*         } else { */
/*                 offset = 0; */
/*                 // print_arr_in_gpu(prefix, size, "dev_prefix"); */
/*                 // printf("-------------- prefix[tid-1] = prefix[%d] = %d\n", tid-1, prefix[tid-1]); */
/*                 // printf("-------------- prefix[tid-2] = prefix[%d] = %d\n", tid-2, prefix[tid-2]); */
/*         } */
/*  */
/*         printf("tid(%d): offset=%d\n", tid, offset); */
/*  */
/*         for (int i=size-1; i>=0; --i) { */
/*                 // if this thread cares for the current number */
/*                 if ((array[i]) == tid) { */
/*                         output[offset++] = array[i]; */
/*                         printf("moving %d from array[%d] to output[%d]\n", array[i], i, offset - 1); */
/*                         // printf("output[%d] = array[%d] = %d\n", offset - 1, i, array[i]); */
/*                 } */
/*         } */
/* } */

int main(void)
{
    int threads = 256;
    int blocks = 2;

    int size = 10;
    elem *array = NULL;
    elem *output = NULL;
    uint *counts = NULL;
    uint *prefix_sums = NULL;

    elem *d_array = NULL;
    uint *d_counts = NULL;
    uint *d_prefix_sums = NULL;
    elem *d_output = NULL;

    array = (elem *)malloc(size * sizeof(elem));
    counts = (uint *)malloc(MAGIC_NUM * sizeof(uint));
    prefix_sums = (uint *)malloc(MAGIC_NUM * sizeof(uint));
    output = (elem *)malloc(size * sizeof(elem));

    cudaErr(hipMalloc((void **)&d_array, size * sizeof(elem)));
    cudaErr(hipMalloc((void **)&d_counts, MAGIC_NUM * sizeof(uint)));
    cudaErr(hipMalloc((void **)&d_output, size * sizeof(elem)));

    for (size_t i = 0; i < size; ++i) {
        // HACK: only because we only iterate once
        array[i] = rand() % MAGIC_NUM;
        output[i] = -1337;
    }
    /* for (int i = 0; i < MAGIC_NUM; ++i) { */
    /*     counts[i] = 0; */
    /* } */

    cudaErr(hipMemcpy(d_array, array, size * sizeof(elem), hipMemcpyHostToDevice));

    // for testing, to make sure the values change
    cudaErr(hipMemcpy(d_output, output, size * sizeof(elem), hipMemcpyHostToDevice));

    print_array(array, size, "unsorted");
    /* test<<<blocks, threads>>>(); */

    countAtomic<<<blocks, threads>>>(d_array, size, d_counts);

    cudaErr(hipMemcpy(counts, d_counts, MAGIC_NUM * sizeof(uint), hipMemcpyDeviceToHost));

    print_array(counts, MAGIC_NUM, "counts");

    d_prefix_sums = prefix_sum(d_counts, MAGIC_NUM, 4, MAGIC_NUM / 4);

    cudaErr(hipMemcpy(prefix_sums, d_prefix_sums, MAGIC_NUM * sizeof(uint), hipMemcpyDeviceToHost));

    print_array(prefix_sums, MAGIC_NUM, "prefix_sum");

    move<<<blocks, threads>>>(d_array, size, d_prefix_sums, d_output);

    cudaErr(hipMemcpy(output, d_output, size * sizeof(elem), hipMemcpyDeviceToHost));

    print_array(output, size, "sorted?");
}
